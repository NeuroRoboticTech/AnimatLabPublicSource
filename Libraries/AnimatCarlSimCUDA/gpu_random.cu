#include "hip/hip_runtime.h"
/*
  Copyright (c) 2007 A. Arnold and J. A. van Meel, FOM institute
  AMOLF, Amsterdam; all rights reserved unless otherwise stated.

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.

  In addition to the regulations of the GNU General Public License,
  publications and communications based in parts on this program or on
  parts of this program are required to cite the article
  "Harvesting graphics power for MD simulations"
  by J.A. van Meel, A. Arnold, D. Frenkel, S. F. Portegies Zwart and
  R. G. Belleman, arXiv:0709.3225.

  This program is distributed in the hope that it will be useful, but
  WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
  General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston,
  MA 02111-1307 USA
*/

#include "gpu.h"
#include "gpu_random.h"

/************************************************
 * GPU kernels
 ************************************************/

/************************************************
 * RNG iteration
 ************************************************/

/** propagate an rand48 RNG one iteration.
    @param Xn  the current RNG state, in 2x 24-bit form
    @param A,C the magic constants for the RNG. For striding,
               this constants have to be adapted, see the constructor
    @result    the new RNG state X(n+1)
*/
__device__
static uint2 RNG_rand48_iterate_single(uint2 Xn, uint2 A, uint2 C)
{
  // results and Xn are 2x 24bit to handle overflows optimally, i.e.
  // in one operation.

  // the multiplication commands however give the low and hi 32 bit,
  // which have to be converted as follows:
  // 48bit in bytes = ABCD EF (space marks 32bit boundary)
  // R0             = ABC
  // R1             =    D EF

  unsigned int R0, R1;

  // low 24-bit multiplication
  const unsigned int lo00 = __umul24(Xn.x, A.x);
  const unsigned int hi00 = __umulhi(Xn.x, A.x);

  // 24bit distribution of 32bit multiplication results
  R0 = (lo00 & 0xFFFFFF);
  R1 = (lo00 >> 24) | (hi00 << 8);

  R0 += C.x; R1 += C.y;

  // transfer overflows
  R1 += (R0 >> 24);
  R0 &= 0xFFFFFF;

  // cross-terms, low/hi 24-bit multiplication
  R1 += __umul24(Xn.y, A.x);
  R1 += __umul24(Xn.x, A.y);

  R1 &= 0xFFFFFF;

  return make_uint2(R0, R1);
}

/************************************************
 * sets of random numbers
 ************************************************/

/** create a set of random numbers. The random numbers are generated in blocks.
    In each block, a thread calculates one random number, the first thread the
    first one, the second the second and so on.
    @param state      the current states of the RNGS, one per thread.
    @param res        where to put the generated numbers
    @param num_blocks how many random numbers each thread generates.
                      The total number of random numbers is therefore
		      num_blocks*nThreads.
    @param A,C        the magic constants for the iteration. They need
                      to be chosen as to advance the RNG by nThreads iterations
		      at once, see the constructor.
*/
__global__
static void RNG_rand48_get_int(uint2 *state, int *res, int num_blocks, uint2 A, uint2 C, int r_max)
{
  const int nThreads = blockDim.x*gridDim.x;

  // load the current state of the RNG into a register
  int   nOutIdx = threadIdx.x + blockIdx.x*blockDim.x;
  uint2 lstate = state[nOutIdx];
  int i;
  for (i = 0; i < num_blocks; ++i) {
    // get upper 31 (!) bits of the 2x 24bits
    res[nOutIdx] = ( lstate.x >> 17 ) | ( lstate.y << 7);
    if(r_max) res[nOutIdx] = res[nOutIdx]%r_max;
    nOutIdx += nThreads;
    // this actually iterates the RNG
    lstate = RNG_rand48_iterate_single(lstate, A, C);
  }

  nOutIdx = threadIdx.x + blockIdx.x*blockDim.x;
  state[nOutIdx] = lstate;
}

/************************************************
 * RNG_rand48 implementation
 ************************************************/

void
RNG_rand48::init(int seed)
{
  // setup execution grid to get max performance
  threadsX = 192;
  blocksX  = 32;

  const int nThreads = threadsX*blocksX;

  uint2* seeds = new uint2[ nThreads ];

  CUDA_CHECK_ERRORS( hipMalloc( (void**) &state, sizeof(uint2)*nThreads ) );

  // calculate strided iteration constants
  unsigned long long A, C;
  A = 1LL; C = 0LL;
  for (int i = 0; i < nThreads; ++i) {
    C += A*c;
    A *= a;
  }
  A0 = (unsigned int)(A & 0xFFFFFFLL);
  A1 = (unsigned int)((A >> 24) & 0xFFFFFFLL);
  C0 = (unsigned int)(C & 0xFFFFFFLL);
  C1 = (unsigned int)((C >> 24) & 0xFFFFFFLL);

  // prepare first nThreads random numbers from seed
  unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
  for (int i = 0; i < nThreads; ++i) {
    x = a*x + c;
    seeds[i].x = (unsigned int)(x & 0xFFFFFFLL);
    seeds[i].y = (unsigned int)((x >> 24) & 0xFFFFFFLL);
  }

  CUDA_CHECK_ERRORS(hipMemcpy(state, seeds, sizeof(uint2)*nThreads, hipMemcpyHostToDevice));

  delete[] seeds;
}

void
RNG_rand48::cleanup() {
  CUDA_CHECK_ERRORS(hipFree((void*) state));
}

void
RNG_rand48::generate_int(int n, int _r_max, int offset)
{
  const int nThreads = threadsX*blocksX;

  int num_blocks = (n + nThreads-1)/nThreads;
	
  if (res == 0) {
    CUDA_CHECK_ERRORS(hipMalloc( (void**) &res, sizeof(int)*nThreads*num_blocks));
  }
  
  dim3 grid( blocksX, 1, 1);
  dim3 threads( threadsX, 1, 1);

  uint2 A, C;
  A.x = A0; A.y = A1;
  C.x = C0; C.y = C1;

  // call GPU kernel
  RNG_rand48_get_int<<< grid, threads >>>((uint2 *)state, (int *)res, num_blocks, A, C, _r_max);
}

// use user specified max value...
void
RNG_rand48::generate(int n, int _r_max)
{
	generate_int(n, _r_max, 0);
}

// use default max value
void
RNG_rand48::generate(int n)
{
	generate_int(n, rand_max, 0);
}

void
RNG_rand48::get(int *r, int n)
 {
  CUDA_CHECK_ERRORS(hipMemcpy( r, res, sizeof(int)*n, hipMemcpyDeviceToHost ) );
}
